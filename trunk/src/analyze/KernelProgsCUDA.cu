
#include <hip/hip_runtime.h>
//! \verbatim
//! ###########################################################################
//! # ASPECT: A spectra clustering tool for exploration of large spectral surveys - Stage I
//! #
//! # Copyright (c) 2009 Aick in der Au
//! # All rights reserved.
//! ###########################################################################
//!
//!      created by : Aick in der Au <aick.inderau@gmail.com>
//!      created on : 12/25/2012
//! additional docs : none
//!  responsibility : 1. Aick in der Au
//!                   2. 
//! \endverbatim
//!
//! \file  KernelProgsCUDA.h
//! \brief CUDA compute functions that are executed on the GPU (or any other CUDA device).


#ifndef EULER
#define EULER 2.718281828f
#endif 

__global__ void clearBMUs( int *_BMUs, int _numNetworkSpectra )
{
	int index = threadIdx.x + blockIdx.x * blockDim.x; 

	if ( index < _numNetworkSpectra )
		_BMUs[index] = -1;
}

__global__ void search( const float *_srcSpectrum, const float *_networkSpectra, float *_outErr, int _numNetworkSpectra, int _numElements)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x; 
	int o = index*_numElements;

	if (index < _numNetworkSpectra)
	{
		float err = 0.0f;
		for (int i=0;i<_numElements;i++)
		{
			float d = _srcSpectrum[i]-_networkSpectra[o+i];
			err += d*d;
		}
		_outErr[index] = err;
	}
}

__global__ void getBMU( float *_errorList, int _numSpectra, int srcSpectraIndex, int *_outBMU )
{
	__syncthreads();

	float err = 100000000000000000000000000000.0;
	int idx = 0;
	for ( int i=0;i<_numSpectra;i++ )
	{
		if ( _errorList[i] < err && _outBMU[i] == -1 ) 
		{
			idx = i;
			err = _errorList[i];
		}
	}

	_outBMU[idx] = srcSpectraIndex;
	_outBMU[_numSpectra] = idx;

	__syncthreads();
}

__global__ void adaptNetwork( 
		const float *spectrum, 
		float *_networkSpectra, 
		int _numElements, 
		int *_outBMU,
		int _gridSize, 
		float _adaptionThreshold, 
		float _sigmaSqr, 
		float _lRate )
{
    const int bestMatchIndex = _outBMU[_numElements];
 	const int xpBestMatch = bestMatchIndex % _gridSize;
 	const int ypBestMatch = bestMatchIndex / _gridSize;
 	const float sigmaSqr2 = _sigmaSqr*(1.f/EULER);
 	const float fGridSizeSqr = _gridSize*_gridSize;


 	// adjust weights of the whole network

 	int x = threadIdx.x + blockIdx.x * blockDim.x; 
	int y = threadIdx.y + blockIdx.y * blockDim.y; 

	if ( x < _gridSize && y < _gridSize )
	{	
		//was: for ( int y=0;y<_gridSize;y++)

 		const float distY1 = y-ypBestMatch;
 		const float distY1Sqr = distY1*distY1;
 		const float distYSqr = distY1Sqr;
 
  		// was: for ( int x=0;x<_gridSize;x++)
  		const float distX1 = x-xpBestMatch;
  		const float distX1Sqr = distX1*distX1;
  		const float distXSqr = distX1Sqr;
  		const float distSqr = (distXSqr+distYSqr)/fGridSizeSqr;					// normalize squared distance with gridsize
  
  		// calculate neighborhood function
  		const float hxy = expf(-sqrtf(distSqr)/sigmaSqr2);						// spike
  		const float lratehsx = _lRate*hxy;
 
 		if ( lratehsx > _adaptionThreshold )
 		{
 			const int spectraAdress = (y*_gridSize+x)*_numElements;
	 		for ( int i=0;i<_numElements;i++ )
 			{
 	 			_networkSpectra[spectraAdress+i] += lratehsx*(spectrum[i]-_networkSpectra[spectraAdress+i]);
   			}
  		}
	}
}

#define NUMTHREADS 512




extern "C" void
process(
	const float *_sourceSpectra, 
	int _numSourceSpectra, 
	float *_networkSpectra, 
	int _numNetworkSpectra, 
	size_t *_pSpectraList, 
	float *_outErr, 
	int *_outBMU, 
	int _numElements, 
	int _gridSize,
	float _adaptionThreshold, 
	float _sigmaSqr, 
	float _lRate )
{

	clearBMUs<<<(_numNetworkSpectra+NUMTHREADS-1)/NUMTHREADS,NUMTHREADS>>>( _outBMU, _numNetworkSpectra );

	// for each input spectrum
	for ( int i=0;i<_numSourceSpectra;i++ ) 
	{
		int srcSpectraIndex = _pSpectraList[i];

		if ( srcSpectraIndex >= 0 || srcSpectraIndex <  _numSourceSpectra)
		{
			int o = srcSpectraIndex*_numElements;

			// calculate euclidean distances to codebook vectors
			search<<<(_numNetworkSpectra+NUMTHREADS-1)/NUMTHREADS,NUMTHREADS>>>( &_sourceSpectra[o], _networkSpectra, _outErr, _numNetworkSpectra, _numElements );
 
 			// calculate best matching unit (BMU) from error list (i.e. euclidean distances)
 			getBMU<<<1,1>>>( _outErr, _numNetworkSpectra, srcSpectraIndex, _outBMU );
 
 			// adapt code book vectors in vicinity.

		    dim3 threadsPerBlock(16, 16);
			dim3 numBlocks((_gridSize+threadsPerBlock.x-1) / threadsPerBlock.x, (_gridSize+threadsPerBlock.y-1) / threadsPerBlock.y);
 			adaptNetwork<<<numBlocks,threadsPerBlock>>>( 
 					&_sourceSpectra[o], 
 					_networkSpectra, 
 					_numElements, 
 					_outBMU, 
 					_gridSize, 
 					_adaptionThreshold, 
 					_sigmaSqr, 
 					_lRate );
		}
	}
}
