
#include <hip/hip_runtime.h>
//! \verbatim
//! ###########################################################################
//! # ASPECT: A spectra clustering tool for exploration of large spectral surveys - Stage I
//! #
//! # Copyright (c) 2009 Aick in der Au
//! # All rights reserved.
//! ###########################################################################
//!
//!      created by : Aick in der Au <aick.inderau@gmail.com>
//!      created on : 12/25/2012
//! additional docs : none
//!  responsibility : 1. Aick in der Au
//!                   2. 
//! \endverbatim
//!
//! \file  KernelProgsCUDA.h
//! \brief CUDA compute functions that are executed on the GPU (or any other CUDA device).


#ifndef EULER
#define EULER 2.718281828f
#endif 

__global__ void clearBMUs( int *_BMUs, int _numNetworkSpectra )
{
	int index = threadIdx.x + blockIdx.x * blockDim.x; 

	if ( index < _numNetworkSpectra )
		_BMUs[index] = -1;
}

__global__ void search( const float *_srcSpectrum, const float *_networkSpectra, float *_outErr, int _numNetworkSpectra, int _numElements)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x; 
	int o = index*_numElements;

	if (index < _numNetworkSpectra)
	{
		float err = 0.0f;
		for (int i=0;i<_numElements;i++)
		{
			float d = _srcSpectrum[i]-_networkSpectra[o+i];
			err += d*d;
		}
		_outErr[index] = err;
	}
}

__global__ void getBMU( float *_errorList, int _numSpectra, int srcSpectraIndex, int *_outBMU )
{
	float err = 3.402823466e+38F;
	int idx = 0;
	for ( int i=0;i<_numSpectra;i++ )
	{
		if ( _errorList[i] < err && _outBMU[i] == -1 ) 
		{
			idx = i;
			err = _errorList[i];
		}
	}

	_outBMU[idx] = srcSpectraIndex;
	_outBMU[_numSpectra] = idx;
}

__global__ void adaptNetwork( 
		const float *spectrum, 
		float *_networkSpectra, 
		int _numElements, 
		int *_outBMU,
		int _gridSize, 
		float _adaptionThreshold, 
		float _sigmaSqr, 
		float _lRate )
{
    const int bestMatchIndex = *_outBMU;
 	const int xpBestMatch = bestMatchIndex % _gridSize;
 	const int ypBestMatch = bestMatchIndex / _gridSize;
 	const float sigmaSqr2 = _sigmaSqr*(1.f/EULER);
 	const float fGridSizeSqr = _gridSize*_gridSize;


 	// adjust weights of the whole network

 	int x = threadIdx.x + blockIdx.x * blockDim.x; 
	int y = threadIdx.y + blockIdx.y * blockDim.y; 

	if ( x < _gridSize && y < _gridSize )
	{	
		//was: for ( int y=0;y<_gridSize;y++)
  		// was: for ( int x=0;x<_gridSize;x++)

  		const float distX1 = x-xpBestMatch;
 		const float distY1 = y-ypBestMatch;
  		const float distXSqr = distX1*distX1;
 		const float distYSqr = distY1*distY1;
  		const float distSqr = (distXSqr+distYSqr)/fGridSizeSqr;					// normalize squared distance with gridsize
  
  		// calculate neighborhood function
  		const float hxy = expf(-sqrtf(distSqr)/sigmaSqr2);						// spike
  		const float lratehsx = _lRate*hxy;
 
 		if ( lratehsx > _adaptionThreshold )
 		{
 			const int spectraAdress = (y*_gridSize+x)*_numElements;
	 		for ( int i=0;i<_numElements;i++ )
 			{
 	 			_networkSpectra[spectraAdress+i] += lratehsx*(spectrum[i]-_networkSpectra[spectraAdress+i]);
   			}
  		}
	}
}

#define NUMTHREADS 512




extern "C" void
process(
	const float *_sourceSpectra, 
	int _numSourceSpectra, 
	float *_networkSpectra, 
	int _numNetworkSpectra, 
	size_t *_pSpectraList, 
	float *_outErr, 
	int *_outBMU, 
	int _numElements, 
	int _gridSize,
	float _adaptionThreshold, 
	float _sigmaSqr, 
	float _lRate )
{

	clearBMUs<<<(_numNetworkSpectra+NUMTHREADS-1)/NUMTHREADS,NUMTHREADS>>>( _outBMU, _numNetworkSpectra );

	hipDeviceSynchronize();

	// for each input spectrum
	for ( int i=0;i<_numSourceSpectra;i++ ) 
	{
		int srcSpectraIndex = _pSpectraList[i];

		if ( srcSpectraIndex >= 0 || srcSpectraIndex <  _numSourceSpectra)
		{
			int o = srcSpectraIndex*_numElements;

			// calculate euclidean distances to codebook vectors
			search<<<(_numNetworkSpectra+NUMTHREADS-1)/NUMTHREADS,NUMTHREADS>>>( &_sourceSpectra[o], _networkSpectra, _outErr, _numNetworkSpectra, _numElements );
			hipDeviceSynchronize();

  			// calculate best matching unit (BMU) from error list (i.e. euclidean distances)
  			getBMU<<<1,1>>>( _outErr, _numNetworkSpectra, srcSpectraIndex, _outBMU );
 			hipDeviceSynchronize();

			// adapt code book vectors in vincinity.
 
 		    dim3 threadsPerBlock(16, 16);
 			dim3 numBlocks((_gridSize+threadsPerBlock.x-1) / threadsPerBlock.x, (_gridSize+threadsPerBlock.y-1) / threadsPerBlock.y);
  			adaptNetwork<<<numBlocks,threadsPerBlock>>>( 
  					&_sourceSpectra[o], 
  					_networkSpectra, 
  					_numElements, 
  					&_outBMU[_numNetworkSpectra], 
  					_gridSize, 
  					_adaptionThreshold, 
  					_sigmaSqr, 
  					_lRate );
			hipDeviceSynchronize();


		}
	}
}
