#include "hip/hip_runtime.h"
//! \verbatim
//! ###########################################################################
//! # ASPECT: A spectra clustering tool for exploration of large spectral surveys - Stage I
//! #
//! # Copyright (c) 2009 Aick in der Au
//! # All rights reserved.
//! ###########################################################################
//!
//!      created by : Aick in der Au <aick.inderau@gmail.com>
//!      created on : 12/25/2012
//! additional docs : none
//!  responsibility : 1. Aick in der Au
//!                   2. 
//! \endverbatim
//!
//! \file  KernelProgsCUDA.h
//! \brief CUDA compute functions that are executed on the GPU (or any other CUDA device).

#include "sdsslib/Timer.h"


#include<stdio.h>

//#include<hip/hip_runtime.h>
//#include<hip/hip_runtime.h>
//#include<hip/hip_runtime_api.h>

#ifndef EULER
#define EULER 2.718281828f
#endif 

#define NUMTHREADS_CLEAR 512
#define NUMBLOCKS_SEARCH 256
#define NUMTHREADS_SEARCHGATHER 512
#define NUMTHREADS_ADAPT 64

__global__ void clearBMUs( int *_BMUs, int _numNetworkSpectra )
{
	int index = threadIdx.x + blockIdx.x * blockDim.x; 

	if ( index < _numNetworkSpectra )
		_BMUs[index] = -1;
}


__global__ void clearErrors( float *_err, int _numNetworkSpectra )
{
	int index = threadIdx.x + blockIdx.x * blockDim.x; 

	if ( index < _numNetworkSpectra )
		_err[index] = 0.0;
}



__global__ void search( const float *_srcSpectrum, const float *_networkSpectra, float *_tempErr, float *_outErr, int _numNetworkSpectra, int _numElements)
{
	int indexDelta = (ceilf((float)_numNetworkSpectra/(float)NUMBLOCKS_SEARCH));
	int indexLow = blockIdx.x*indexDelta; 
	int indexHigh = indexLow + indexDelta;
	int thrdIdx = threadIdx.x;

	__shared__ float err[16536];

	indexLow = min( indexLow, _numNetworkSpectra-1 );
	indexHigh = min( indexHigh, _numNetworkSpectra );

	int nTotalThreads = blockDim.x;

	while(nTotalThreads > 1)
	{
		int halfPoint = (nTotalThreads >> 1);	

		if (threadIdx.x < halfPoint)
		{
			for ( int index = indexLow;index<indexHigh;index++)
			{
				int thread2 = threadIdx.x + halfPoint;

				int il = index-indexLow;
				int o = index*_numElements;
				float d1 = _srcSpectrum[thrdIdx]-_networkSpectra[o+thrdIdx];
				float d2 = _srcSpectrum[thread2]-_networkSpectra[o+thread2];

				err[il*blockDim.x+thrdIdx] = d1*d1+d2*d2;
			}
		}

		__syncthreads();

	}


	for ( int index = indexLow;index<indexHigh;index++)
	{
		int il = index-indexLow;
		_outErr[index] = err[il*blockDim.x];
	}
}

__global__ void gatherSearch( float *_errorList, int _numSpectra, int srcSpectraIndex, int *_outBMU )
{
	__shared__ float t_err[NUMTHREADS_SEARCHGATHER];
	__shared__ int t_erridx[NUMTHREADS_SEARCHGATHER];

	int index = threadIdx.x;
	int nWidth = _numSpectra/NUMTHREADS_SEARCHGATHER;
	int nLow = nWidth*threadIdx.x;
	int nHigh = nLow+nWidth;
	if ( nHigh >= _numSpectra ) 
		nHigh = _numSpectra-1;

	float err = 3.402823466e+38F;
	int erridx;
	for ( int i=nLow;i<nHigh;i++ )
	{
		if ( _errorList[i] < err && _outBMU[i] == -1 ) 
		{
			erridx = i;
			err = _errorList[i];
		}
	}

	t_err[index] = err;
	t_erridx[index] = erridx;

	__syncthreads();

	// gather all per thread search results 
	if ( index == 0)
	{
		err = t_erridx[0];
		erridx = t_erridx[0];

		for ( int i=1;i<NUMTHREADS_SEARCHGATHER;i++ )
		{
			if ( t_err[i] < err ) 
			{
				erridx = t_erridx[i];
				err = t_err[i];
			}

		}
		_outBMU[erridx] = srcSpectraIndex;
		_outBMU[_numSpectra] = erridx;
	}
}
/*
// Reduction (min/max/avr/sum), valid only when blockDim.x is a power of two:
int  thread2;
double temp;
__shared__ double min[BLOCK_SIZE], max[BLOCK_SIZE], avg[BLOCK_SIZE], sum[BLOCK_SIZE];

int nTotalThreads = blockDim.x;	// Total number of active threads

while(nTotalThreads > 1)
{
	int halfPoint = (nTotalThreads >> 1);	// divide by two
	// only the first half of the threads will be active.

	if (threadIdx.x < halfPoint)
	{
		thread2 = threadIdx.x + halfPoint;

		// Get the shared value stored by another thread
		temp = min[thread2];
		if (temp < min[threadIdx.x]) 
			min[threadIdx.x] = temp; 

		temp = max[thread2];
		if (temp > max[threadIdx.x]) 
			max[threadIdx.x] = temp;			

		sum[threadIdx.x] += sum[thread2];

		// when calculating the average, sum and divide
		avg[threadIdx.x] += avg[thread2];
		avg[threadIdx.x] /= 2;
	}
	__syncthreads();

	// Reducing the binary tree size by two:
	nTotalThreads = halfPoint;
}
*/

__global__ void adaptNetwork( 
		const float *spectrum, 
		float *_networkSpectra, 
		int _numElements, 
		int *_outBMU,
		int _gridSize, 
		float _adaptionThreshold, 
		float _sigmaSqr, 
		float _lRate )
{
    const int bestMatchIndex = *_outBMU;
 	const int xpBestMatch = bestMatchIndex % _gridSize;
 	const int ypBestMatch = bestMatchIndex / _gridSize;
 	const float sigmaSqr2 = _sigmaSqr*(1.f/EULER);
 	const float fGridSizeSqr = _gridSize*_gridSize;


 	// adjust weights of the whole network

 	int x = threadIdx.x + blockIdx.x * blockDim.x; 
	int y = threadIdx.y + blockIdx.y * blockDim.y; 

	if ( x < _gridSize && y < _gridSize )
	{	
		//was: for ( int y=0;y<_gridSize;y++)
  		//was: for ( int x=0;x<_gridSize;x++)

  		const float distX1 = x-xpBestMatch;
 		const float distY1 = y-ypBestMatch;
  		const float distXSqr = distX1*distX1;
 		const float distYSqr = distY1*distY1;
  		const float distSqr = (distXSqr+distYSqr)/fGridSizeSqr;					// normalize squared distance with gridsize
  
  		// calculate neighborhood function
  		const float hxy = expf(-sqrtf(distSqr)/sigmaSqr2);						// spike
  		const float lratehsx = _lRate*hxy;
 
 		if ( lratehsx > _adaptionThreshold )
 		{
 			const int spectraAdress = (y*_gridSize+x)*_numElements;
	 		for ( int i=0;i<_numElements;i++ )
 			{
 	 			_networkSpectra[spectraAdress+i] += lratehsx*(spectrum[i]-_networkSpectra[spectraAdress+i]);
   			}
  		}
	}
}



extern "C" void
process(
	const float *_sourceSpectra, 
	int _numSourceSpectra, 
	float *_networkSpectra, 
	int _numNetworkSpectra, 
	size_t *_pSpectraList, 
	float *_tempErr, 
	float *_outErr, 
	int *_outBMU, 
	int _numElements, 
	int _gridSize,
	float _adaptionThreshold, 
	float _sigmaSqr, 
	float _lRate )
{

	Timer t;

	double tSearch = 0.0;
	double tSearchCollect = 0.0;
	double tAdapt = 0.0;

	clearBMUs<<<(_numNetworkSpectra+NUMTHREADS_CLEAR-1)/NUMTHREADS_CLEAR,NUMTHREADS_CLEAR>>>( _outBMU, _numNetworkSpectra );

	hipDeviceSynchronize();

	// for each input spectrum
	for ( int i=0;i<_numSourceSpectra;i++ ) 
	{
		int srcSpectraIndex = _pSpectraList[i];

		if ( srcSpectraIndex >= 0 || srcSpectraIndex <  _numSourceSpectra)
		{
			int o = srcSpectraIndex*_numElements;
			Timer t2;

			// calculate euclidean distances to codebook vectors
			clearErrors<<<(_numNetworkSpectra+NUMTHREADS_CLEAR-1)/NUMTHREADS_CLEAR,NUMTHREADS_CLEAR>>>( _outErr, _numNetworkSpectra );
			search<<<NUMBLOCKS_SEARCH,_numElements>>>( &_sourceSpectra[o], _networkSpectra, _tempErr, _outErr, _numNetworkSpectra, _numElements );
			hipDeviceSynchronize();
 		
			tSearch += t2.getElapsedSecs();
			t2.start();

			// calculate best matching unit (BMU) from error list (i.e. euclidean distances)
			
  			gatherSearch<<<1,NUMTHREADS_SEARCHGATHER>>>( _outErr, _numNetworkSpectra, srcSpectraIndex, _outBMU );
 			hipDeviceSynchronize();

			tSearchCollect += t2.getElapsedSecs();
			t2.start();
  
			// adapt code book vectors in vincinity.
 
 		    dim3 threadsPerBlock(NUMTHREADS_ADAPT, NUMTHREADS_ADAPT);
 			dim3 numBlocks((_gridSize+threadsPerBlock.x-1) / threadsPerBlock.x, (_gridSize+threadsPerBlock.y-1) / threadsPerBlock.y);
  			adaptNetwork<<<numBlocks,threadsPerBlock>>>( 
  					&_sourceSpectra[o], 
  					_networkSpectra, 
  					_numElements, 
  					&_outBMU[_numNetworkSpectra], 
  					_gridSize, 
  					_adaptionThreshold, 
  					_sigmaSqr, 
  					_lRate );
			hipDeviceSynchronize();

			tAdapt += t2.getElapsedSecs();
		}
	}
	double dt = t.getElapsedSecs();

	printf("\n\n%f secs search\n", tSearch);
	printf("%f secs search collect\n", tSearchCollect);
	printf("%f secs adapt\n", tAdapt);
	printf("%f secs total\n\n", dt);
}
